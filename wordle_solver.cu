#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <algorithm>
#include <vector>
#include <string>
#include <fstream>
#include <iostream>
#include <chrono>

#define WORD_LENGTH 5
#define MAX_WORDS 15000
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(1); \
        } \
    } while(0)

// Device function to calculate hint for a guess-answer pair
__device__ unsigned char getHint(const char* guess, const char* answer) {
    unsigned char charHints[5] = {0};
    
    // Check for exact matches first
    for (int i = 0; i < 5; i++) {
        if (guess[i] == answer[i]) {
            charHints[i] = 2; // Green (correct position)
        }
    }
    
    // Check for wrong position matches
    for (int i = 0; i < 5; i++) {
        if (charHints[i] == 0) { // Not already green
            for (int j = 0; j < 5; j++) {
                if (i != j && charHints[j] != 2 && guess[i] == answer[j]) {
                    // Make sure this letter isn't already accounted for
                    bool alreadyUsed = false;
                    for (int k = 0; k < i; k++) {
                        if (charHints[k] == 1 && guess[k] == guess[i]) {
                            alreadyUsed = true;
                            break;
                        }
                    }
                    if (!alreadyUsed) {
                        charHints[i] = 1; // Yellow (wrong position)
                        break;
                    }
                }
            }
        }
    }
    
    // Convert to single hint value (base 3)
    unsigned char ret = 0;
    for (int i = 0; i < 5; i++) {
        ret = (ret * 3) + charHints[i];
    }
    
    return ret;
}

// Kernel to calculate all hints for all guess-answer pairs
__global__ void calculateAllHints(char* guesses, char* answers, unsigned char* hints, 
                                  int numGuesses, int numAnswers) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = numGuesses * numAnswers;
    
    if (idx < total) {
        int guessIdx = idx / numAnswers;
        int answerIdx = idx % numAnswers;
        
        char* guess = guesses + guessIdx * (WORD_LENGTH + 1);
        char* answer = answers + answerIdx * (WORD_LENGTH + 1);
        
        hints[idx] = getHint(guess, answer);
    }
}

// Device function to count matching answers for a hint
__device__ int countMatchingAnswers(char* guesses, char* answers, unsigned char* hints,
                                   int guessIdx, unsigned char targetHint, int numAnswers) {
    int count = 0;
    for (int i = 0; i < numAnswers; i++) {
        int hintIdx = guessIdx * numAnswers + i;
        if (hints[hintIdx] == targetHint) {
            count++;
        }
    }
    return count;
}

// Device function to calculate average number of candidates for a guess pair
__device__ float avgNumCandidates(char* guesses, char* answers, unsigned char* hints,
                                 int guess1Idx, int guess2Idx, int numAnswers) {
    float total = 0.0f;
    
    for (int answerIdx = 0; answerIdx < numAnswers; answerIdx++) {
        // Get hint for first guess against this answer
        unsigned char hint1 = hints[guess1Idx * numAnswers + answerIdx];
        
        // Count how many answers match this hint for first guess
        int candidates = countMatchingAnswers(guesses, answers, hints, guess1Idx, hint1, numAnswers);
        
        if (candidates <= 2) {
            total += 1.0f;
        } else {
            // Apply second guess to remaining candidates
            int finalCandidates = 0;
            for (int i = 0; i < numAnswers; i++) {
                if (hints[guess1Idx * numAnswers + i] == hint1) {
                    unsigned char hint2 = hints[guess2Idx * numAnswers + i];
                    // Count how many of the remaining candidates match hint2
                    for (int j = 0; j < numAnswers; j++) {
                        if (hints[guess1Idx * numAnswers + j] == hint1 && 
                            hints[guess2Idx * numAnswers + j] == hint2) {
                            finalCandidates++;
                        }
                    }
                    break; // We only need to do this calculation once per unique hint1
                }
            }
            total += (float)finalCandidates;
        }
    }
    
    return total / (float)numAnswers;
}

// Kernel to find best guess pairs
__global__ void findBestGuessPairs(char* guesses, char* answers, unsigned char* hints,
                                  int* filteredIndices, float* results, int numFiltered, int numAnswers) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalPairs = numFiltered * (numFiltered - 1) / 2;
    
    if (idx < totalPairs) {
        // Convert linear index to i,j pair indices
        int i = 0;
        int remaining = idx;
        while (remaining >= (numFiltered - 1 - i)) {
            remaining -= (numFiltered - 1 - i);
            i++;
        }
        int j = i + 1 + remaining;
        
        int guess1Idx = filteredIndices[i];
        int guess2Idx = filteredIndices[j];
        
        // Check if guesses share letters (skip if they do)
        char* guess1 = guesses + guess1Idx * (WORD_LENGTH + 1);
        char* guess2 = guesses + guess2Idx * (WORD_LENGTH + 1);
        
        bool shareLetters = false;
        for (int a = 0; a < 5; a++) {
            for (int b = 0; b < 5; b++) {
                if (guess1[a] == guess2[b]) {
                    shareLetters = true;
                    break;
                }
            }
            if (shareLetters) break;
        }
        
        if (!shareLetters) {
            results[idx] = avgNumCandidates(guesses, answers, hints, guess1Idx, guess2Idx, numAnswers);
        } else {
            results[idx] = 999999.0f; // Large value to indicate invalid pair
        }
    }
}

// Host function to load words from file
std::vector<std::string> loadWords(const char* filename) {
    std::vector<std::string> words;
    std::ifstream file(filename);
    std::string line;
    
    while (std::getline(file, line)) {
        if (line.length() == WORD_LENGTH) {
            words.push_back(line);
        }
    }
    
    return words;
}

// Host function to check if word has 5 unique letters
bool hasUniqueLetters(const std::string& word) {
    bool seen[26] = {false};
    for (char c : word) {
        int idx = c - 'a';
        if (seen[idx]) return false;
        seen[idx] = true;
    }
    return true;
}

int main() {
    auto start = std::chrono::high_resolution_clock::now();
    
    // Load word lists
    std::vector<std::string> guessesVec = loadWords("io/guesses.txt");
    std::vector<std::string> answersVec = loadWords("io/answers.txt");
    
    int numGuesses = guessesVec.size();
    int numAnswers = answersVec.size();
    
    printf("Loaded %d guesses and %d answers\n", numGuesses, numAnswers);
    
    // Allocate host memory
    char* h_guesses = (char*)malloc(numGuesses * (WORD_LENGTH + 1) * sizeof(char));
    char* h_answers = (char*)malloc(numAnswers * (WORD_LENGTH + 1) * sizeof(char));
    
    // Copy words to host arrays
    for (int i = 0; i < numGuesses; i++) {
        strcpy(h_guesses + i * (WORD_LENGTH + 1), guessesVec[i].c_str());
    }
    for (int i = 0; i < numAnswers; i++) {
        strcpy(h_answers + i * (WORD_LENGTH + 1), answersVec[i].c_str());
    }
    
    // Allocate device memory
    char* d_guesses;
    char* d_answers;
    unsigned char* d_hints;
    
    CUDA_CHECK(hipMalloc(&d_guesses, numGuesses * (WORD_LENGTH + 1) * sizeof(char)));
    CUDA_CHECK(hipMalloc(&d_answers, numAnswers * (WORD_LENGTH + 1) * sizeof(char)));
    CUDA_CHECK(hipMalloc(&d_hints, numGuesses * numAnswers * sizeof(unsigned char)));
    
    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_guesses, h_guesses, numGuesses * (WORD_LENGTH + 1) * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_answers, h_answers, numAnswers * (WORD_LENGTH + 1) * sizeof(char), hipMemcpyHostToDevice));
    
    // Calculate hints on GPU
    printf("Calculating hints on GPU...\n");
    int totalHints = numGuesses * numAnswers;
    int blockSize = 256;
    int gridSize = (totalHints + blockSize - 1) / blockSize;
    
    calculateAllHints<<<gridSize, blockSize>>>(d_guesses, d_answers, d_hints, numGuesses, numAnswers);
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Filter guesses with 5 unique letters
    std::vector<int> filteredIndices;
    for (int i = 0; i < numGuesses; i++) {
        if (hasUniqueLetters(guessesVec[i])) {
            filteredIndices.push_back(i);
        }
    }
    
    int numFiltered = filteredIndices.size();
    int totalPairs = numFiltered * (numFiltered - 1) / 2;
    
    printf("Filtered to %d guesses with unique letters (%d pairs)\n", numFiltered, totalPairs);
    
    // Allocate memory for filtered indices and results
    int* d_filteredIndices;
    float* d_results;
    
    CUDA_CHECK(hipMalloc(&d_filteredIndices, numFiltered * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_results, totalPairs * sizeof(float)));
    
    CUDA_CHECK(hipMemcpy(d_filteredIndices, filteredIndices.data(), numFiltered * sizeof(int), hipMemcpyHostToDevice));
    
    // Find best guess pairs on GPU
    printf("Finding best guess pairs on GPU...\n");
    gridSize = (totalPairs + blockSize - 1) / blockSize;
    
    findBestGuessPairs<<<gridSize, blockSize>>>(d_guesses, d_answers, d_hints, d_filteredIndices, d_results, numFiltered, numAnswers);
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy results back and find minimum
    float* h_results = (float*)malloc(totalPairs * sizeof(float));
    CUDA_CHECK(hipMemcpy(h_results, d_results, totalPairs * sizeof(float), hipMemcpyDeviceToHost));
    
    float bestScore = 999999.0f;
    int bestIdx = -1;
    
    for (int i = 0; i < totalPairs; i++) {
        if (h_results[i] < bestScore) {
            bestScore = h_results[i];
            bestIdx = i;
        }
    }
    
    // Convert best index back to guess pair
    int i = 0, remaining = bestIdx;
    while (remaining >= (numFiltered - 1 - i)) {
        remaining -= (numFiltered - 1 - i);
        i++;
    }
    int j = i + 1 + remaining;
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    
    printf("\nBest guess pair: %s, %s (score: %.2f)\n", 
           guessesVec[filteredIndices[i]].c_str(), 
           guessesVec[filteredIndices[j]].c_str(), 
           bestScore);
    printf("Total execution time: %ld ms\n", duration.count());
    
    // Cleanup
    free(h_guesses);
    free(h_answers);
    free(h_results);
    hipFree(d_guesses);
    hipFree(d_answers);
    hipFree(d_hints);
    hipFree(d_filteredIndices);
    hipFree(d_results);
    
    return 0;
} 